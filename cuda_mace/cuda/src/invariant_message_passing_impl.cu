#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include "invariant_message_passing_impl.cuh"
#include "torch_utils.cuh"

#include <iostream>
#include <torch/script.h>

using namespace std;
using namespace torch::indexing;
using namespace torch::autograd;

#define CHECK_CUDA(x)                                                          \
  TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x)                                                    \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x)                                                         \
  CHECK_CUDA(x);                                                               \
  CHECK_CONTIGUOUS(x)

#define FULL_MASK 0xffffffff

#define WARP_SIZE 32
#define NWARPS_PER_BLOCK 4
#define NEIGHBOUR_NEDGES_PER_BLOCK 512

template <typename scalar_t, const int TM, const int TN>
__global__ void inv_tp_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        X, // [nnodes nchannels]
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        Y, // [nedges, (L+1)**2]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits>
        radial,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        sender_list, //
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        receiver_list, // which index we need to sum a particular edge into ->
                       // monotonically increasing.
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        first_occurences, // the indexes in reciever_list which deliniate the
                          // set of edges per node.
    torch::PackedTensorAccessor64<int32_t, 2, torch::RestrictPtrTraits>
        node_edge_index,
    torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits>
        output) {

  extern __shared__ char buffer[];

  void *sptr = buffer;
  unsigned int space = 0;

  int32_t *buffer_sender = shared_array<int32_t>(512, sptr, &space);

  const uint threadCol = threadIdx.x % WARP_SIZE;
  const uint threadRow = threadIdx.x / WARP_SIZE;

  const uint N = X.size(1);
  const uint edge_start = first_occurences[blockIdx.x];
  const uint edge_end = first_occurences[output.size(0) + blockIdx.x];
  const uint node_index = receiver_list[edge_start];

  scalar_t regY[TM] = {0.0};
  scalar_t regX[TN] = {0.0};
  scalar_t regRadial[TM * TN] = {0.0};
  scalar_t regOut[TN * TM] = {0.0};
  scalar_t regC[TN * TM] = {0.0};
  // check if this node has neighbours
  if (edge_end - edge_start == 0) {
    return;
  }

  for (int tid = threadIdx.x; tid < edge_end - edge_start; tid += blockDim.x) {
    buffer_sender[tid] = sender_list[edge_start + tid];
  }

  __syncthreads();

  for (int edge = edge_start + threadIdx.x; edge < edge_end;
       edge += blockDim.x) {
    int32_t sender = buffer_sender[edge - edge_start];
    node_edge_index[sender][node_index] = edge;
  }

  for (int feature = threadCol; feature < N; feature += WARP_SIZE * TN) {
    for (int m = threadRow; m < 16; m += NWARPS_PER_BLOCK * TM) {

      for (int i = 0; i < TN; i++) {
        for (int j = 0; j < TM; j++) {
          regC[i * TM + j] = 0.0;
          regOut[i * TM + j] = 0.0;
        }
      }

      for (uint edge = edge_start; edge < edge_end; edge++) {
        for (int i = 0; i < TN; i++) {
          regX[i] =
              X[buffer_sender[edge - edge_start]][i * WARP_SIZE + feature];
        }
        for (int i = 0; i < TN; i++) {
          for (int j = 0; j < TM; j++) {
            int32_t lm_index = sqrt(j * NWARPS_PER_BLOCK + m);

            regRadial[i * TM + j] =
                radial[edge][lm_index][i * WARP_SIZE + feature];
          }
        }

        for (int j = 0; j < TM; j++) {
          regY[j] = Y[j * NWARPS_PER_BLOCK + m][edge];
        }

        for (int i = 0; i < TN; i++) {
          for (int j = 0; j < TM; j++) {
            scalar_t val = regX[i] * regY[j] * regRadial[i * TM + j];
            scalar_t val_compensated = val - regC[i * TM + j];
            scalar_t tmp_new = regOut[i * TM + j] + val_compensated;
            regC[i * TM + j] = (tmp_new - regOut[i * TM + j]) - val_compensated;
            regOut[i * TM + j] = tmp_new;
          }
        }
      }

      __syncthreads();

      for (int i = 0; i < TN; i++) {
        for (int j = 0; j < TM; j++) {
          output[node_index][j * NWARPS_PER_BLOCK + m]
                [i * WARP_SIZE + feature] = regOut[i * TM + j];
        }
      }
    }
  }
}

std::vector<torch::Tensor>
forward_gpu(torch::Tensor X, torch::Tensor Y, torch::Tensor radial,
            torch::Tensor sender_list, torch::Tensor receiver_list,
            torch::Tensor first_occurences, const int64_t nnodes) {
  const uint nedges = Y.size(1);
  const int nspherical_harm = Y.size(0);
  const int nfeatures = X.size(1);

  TORCH_CHECK(nfeatures % WARP_SIZE == 0,
              "feature dimension must be a multiple of 32");
  TORCH_CHECK(nspherical_harm == 16,
              "number of edge spherical harmonics must be 16");
  TORCH_CHECK(nfeatures <= 128, "feature dimension cannot be greater than 128");

  torch::Tensor node_edge_index = torch::empty(
      {nnodes, nnodes},
      torch::TensorOptions().dtype(torch::kInt32).device(X.device()));

  torch::Tensor output =
      torch::empty({nnodes, nspherical_harm, nfeatures},
                   torch::TensorOptions().dtype(X.dtype()).device(X.device()));

  dim3 gridDim(nnodes);

  dim3 blockDim(NWARPS_PER_BLOCK * WARP_SIZE, 1, 1);

  AT_DISPATCH_FLOATING_TYPES(
      X.type(), "forward_gpu",
      ([&] {
        unsigned int space = 0;
        void *sptr;
        shared_array<int32_t>(512, sptr, &space);
        // shared_array<int32_t>(512, sptr, &space);

        if (nfeatures >= 128) {
          inv_tp_kernel<scalar_t, 4, 4><<<gridDim, blockDim, space>>>(
              X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              node_edge_index
                  .packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>(),
              output
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
        } else if (nfeatures == 96) {
          inv_tp_kernel<scalar_t, 4, 3><<<gridDim, blockDim, space>>>(
              X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              node_edge_index
                  .packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>(),
              output
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
        } else if (nfeatures == 64) {
          inv_tp_kernel<scalar_t, 4, 2><<<gridDim, blockDim, space>>>(
              X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              node_edge_index
                  .packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>(),
              output
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
        } else if (nfeatures == 32) {

          inv_tp_kernel<scalar_t, 4, 1><<<gridDim, blockDim, space>>>(
              X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              node_edge_index
                  .packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>(),
              output
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
        }
      }

       ));

  return {output, node_edge_index};
}

template <typename scalar_t, const int TM, const int TN>
__global__ void backward_edge_inv_tp_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        X, // [nedges, feat]
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        Y, // [nedges, m]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits>
        radial, // [nedges, LMAX, feat]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits>
        grad_in, // [nnodes, m, feat]
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        sender_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        receiver_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        first_occurences,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> gradY,
    torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits>
        grad_radial) {
  extern __shared__ char buffer[];

  void *sptr = buffer;
  unsigned int space = 0;

  scalar_t *buffer_grad_in =
      shared_array<scalar_t>(16 * X.size(1), sptr, &space);
  scalar_t *buffer_Y =
      shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr, &space);
  scalar_t *buffer_dY =
      shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr, &space);

  const uint threadCol = threadIdx.x % WARP_SIZE;
  const uint threadRow = threadIdx.x / WARP_SIZE;

  const uint edge_start = first_occurences[blockIdx.x];
  const uint node_index = receiver_list[edge_start];
  const uint edge_end = first_occurences[grad_in.size(0) + blockIdx.x];

  const uint N_start = blockIdx.y * TN * WARP_SIZE;

  scalar_t regX[TN] = {0.0};
  scalar_t regW[4 * TN] = {0.0};

  scalar_t regGradW[4 * TN] = {0.0};

  if (edge_end - edge_start == 0) {
    return;
  }

  for (int m = 0; m < 16 / NWARPS_PER_BLOCK; m++) {
    for (int n = 0; n < TN; n++) {
      // if (N_start + n * WARP_SIZE + threadCol < X.size(1))
      buffer_grad_in[(m * NWARPS_PER_BLOCK + threadRow) * X.size(1) +
                     n * WARP_SIZE + threadCol] =
          grad_in[node_index][m * NWARPS_PER_BLOCK + threadRow]
                 [N_start + n * WARP_SIZE + threadCol];
    }
  }

  __syncthreads();

  int niter = find_integer_divisor(edge_end - edge_start, NWARPS_PER_BLOCK);

  for (uint ni = 0; ni < niter; ni++) {
    uint edge = edge_start + ni * NWARPS_PER_BLOCK + threadRow;

    if (edge < edge_end) {
      uint sender_id = sender_list[edge];

      if (threadCol < 16) {
        buffer_Y[threadCol * NWARPS_PER_BLOCK + threadRow] = Y[threadCol][edge];
        buffer_dY[threadCol * NWARPS_PER_BLOCK + threadRow] = 0.0;
      }

      __syncwarp();

      for (int n = 0; n < TN; n++) {

        regX[n] = X[sender_id][N_start + n * WARP_SIZE + threadCol];
      }

      for (int n = 0; n < TN; n++) {
        for (int L = 0; L < 4; L++) {
          regGradW[L * TN + n] = 0.0;
          regW[L * TN + n] =
              radial[edge][L][N_start + n * WARP_SIZE + threadCol];
        }
      }

      for (int L = 0; L < 4; L++) {
        uint mstart = L * L;
        uint mend = (L + 1) * (L + 1);

        for (int m = mstart; m < mend; m++) {
          scalar_t sph =
              buffer_Y[m * NWARPS_PER_BLOCK + threadRow]; // Y[edge][m];

          scalar_t dgradY = 0.0;

          for (int n = 0; n < TN; n++) {
            //  scalar_t gradin = regGradIn[m * TN + n];
            scalar_t gradin =
                buffer_grad_in[m * X.size(1) + n * WARP_SIZE + threadCol];
            scalar_t w = regW[L * TN + n];

            regGradW[L * TN + n] += sph * regX[n] * gradin;

            dgradY += gradin * w * regX[n];
          }

          for (int offset = 16; offset > 0; offset /= 2) {
            dgradY += __shfl_down_sync(FULL_MASK, dgradY, offset, WARP_SIZE);
          }

          // threadIdx % WARP_SIZE = 0 dgradY contains the derivative of the
          // output wrt. Y
          if (threadCol == 0)
            buffer_dY[m * NWARPS_PER_BLOCK + threadRow] = dgradY;
        }
      }
    }

    __syncthreads();

    if (edge < edge_end) {
      if (threadCol < 16) {
        gradY[threadCol][edge] =
            buffer_dY[threadCol * NWARPS_PER_BLOCK + threadRow];
      }

      for (int n = 0; n < TN; n++) {
        //  write grad_radial
        for (int L = 0; L < 4; L++) {
          grad_radial[edge][L][N_start + n * WARP_SIZE + threadCol] =
              regGradW[L * TN + n];
        }
      }
    }
  }
}

template <typename scalar_t, const int TM, const int TN>
__global__ void backward_node_inv_tp_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        Y,
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits>
        radial,
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits>
        grad_in,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        sender_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        receiver_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits>
        first_occurences,
    const torch::PackedTensorAccessor64<int32_t, 2, torch::RestrictPtrTraits>
        node_edge_index,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        gradX) {

  extern __shared__ char buffer[];

  void *sptr = buffer;
  unsigned int space = 0;

  scalar_t regY[TM] = {0.0};
  scalar_t regRadial[TM * TN] = {0.0};
  scalar_t regGradIn[TM * TN] = {0.0};
  scalar_t regGradX[TN] = {0.0};
  scalar_t regGradXC[TN] = {0.0};

  scalar_t *buffer_out =
      shared_array<scalar_t>(NWARPS_PER_BLOCK * WARP_SIZE, sptr, &space);
  int32_t *buffer_sorted_sender_idx = shared_array<int32_t>(512, sptr, &space);
  int32_t *buffer_receiver_list = shared_array<int32_t>(512, sptr, &space);

  const uint threadCol = threadIdx.x % WARP_SIZE;
  const uint threadRow = threadIdx.x / WARP_SIZE;

  const uint edge_start = first_occurences[blockIdx.x];
  // const uint node_index = sender_list[sorted_sender_idx[edge_start]];
  const uint node_index = receiver_list[edge_start];
  const uint edge_end = first_occurences[grad_in.size(0) + blockIdx.x];

  if (edge_end - edge_start == 0) {
    return;
  }

  for (int tid = threadIdx.x; tid < edge_end - edge_start; tid += blockDim.x) {
    int sender = sender_list[edge_start + tid];
    int sorted_id = node_edge_index[node_index][sender];
    buffer_sorted_sender_idx[tid] = sorted_id;
    buffer_receiver_list[tid] = receiver_list[sorted_id];
  }

  __syncthreads();

  /*
  edge, sender, receiver, sorted_sender_by_sender, sorted_receiver_by_sender
  0 |27 0 0 1
  1 |45 0 0 27
  2 |55 0 0 45
  3 |1 0 0 55
  4 |0 1 1 0
  5 |2 1 1 2
  6 |4 1 1 4
  7 |6 1 1 6

  block: 0, node_index: 0 edge: 0, receiver_id: 1 sorted_id :4
  block: 0, node_index: 0 edge: 1, receiver_id: 27 sorted_id :111
  block: 0, node_index: 0 edge: 2, receiver_id: 45 sorted_id :183
  block: 0, node_index: 0 edge: 3, receiver_id: 55 sorted_id :223

  block: 1, node_index: 1 edge: 4, receiver_id: 0 sorted_id :3
  block: 1, node_index: 1 edge: 6, receiver_id: 4 sorted_id :17
  block: 1, node_index: 1 edge: 7, receiver_id: 6 sorted_id :26
  */
  for (int feature = threadCol; feature < gradX.size(1);
       feature += WARP_SIZE * TN) {
    __syncthreads();

    for (int i = 0; i < TN; i++) {
      regGradX[i] = 0.0;
      regGradXC[i] = 0.0;
    }

    for (int32_t edge = edge_start; edge < edge_end; edge++) {

      int32_t sorted_id = buffer_sorted_sender_idx[edge - edge_start];
      int32_t receiver_id = buffer_receiver_list[edge - edge_start];

      for (int m = threadRow; m < 16; m += NWARPS_PER_BLOCK * TM) {
        for (int j = 0; j < TM; j++) {
          regY[j] = Y[j * NWARPS_PER_BLOCK + m][sorted_id];
          // regY[j] = Y[edge][j * NWARPS_PER_BLOCK + m];
        }

        for (int j = 0; j < TM; j++) {
          int32_t lm_index = sqrt(j * NWARPS_PER_BLOCK + m);
          for (int i = 0; i < TN; i++) {
            regRadial[i * TM + j] =
                radial[sorted_id][lm_index][i * WARP_SIZE + feature];
            // regRadial[i * TM + j] = radial[edge][lm_index][i * WARP_SIZE +
            // feature];
            regGradIn[i * TM + j] =
                grad_in[receiver_id][j * NWARPS_PER_BLOCK + m]
                       [i * WARP_SIZE + feature];
            // regGradIn[i * TM + j] = grad_in[buffer_sender[edge -
            // edge_start]][j * NWARPS_PER_BLOCK + m][i * WARP_SIZE + feature];
          }
        }

        for (int i = 0; i < TN; i++) {
          for (int j = 0; j < TM; j++) {
            scalar_t val =
                regGradIn[i * TM + j] * regRadial[i * TM + j] * regY[j];
            scalar_t val_compensated = val - regGradXC[i];
            scalar_t tmp_new = regGradX[i] + val_compensated;
            regGradXC[i] = (tmp_new - regGradX[i]) - val_compensated;
            regGradX[i] = tmp_new;
          }
        }
      }
    }

    for (int i = 0; i < TN; i++) {
      __syncthreads();

      buffer_out[threadRow * WARP_SIZE + threadCol] = regGradX[i];

      __syncthreads();
      /* need to reduce over m here*/
      if (threadRow == 0) {
        scalar_t tmp = 0.0;
        for (int j = 0; j < NWARPS_PER_BLOCK; j++) {
          tmp += buffer_out[j * WARP_SIZE + threadCol];
        }

        gradX[node_index][i * WARP_SIZE + feature] = tmp;
      }
    }
  }
}

std::vector<torch::Tensor>
backward_gpu(torch::Tensor X, torch::Tensor Y, torch::Tensor radial,
             torch::Tensor grad_in, torch::Tensor sender_list,
             torch::Tensor receiver_list, torch::Tensor first_occurences,
             torch::Tensor node_edge_index, const int64_t nnodes) {
  uint nedges = Y.size(1);
  uint nfeatures = X.size(1);

  TORCH_CHECK(X.requires_grad(), "X must require grad for invariant message "
                                 "passing backwards_kernel to be called.");
  TORCH_CHECK(Y.requires_grad(), "Y must require grad for invariant message "
                                 "passing backwards_kernel to be called.");
  TORCH_CHECK(radial.requires_grad(),
              "radial must require grad for invariant message passing "
              "backwards_kernel to be called.");

  torch::Tensor gradRadial = torch::empty_like(
      radial,
      torch::TensorOptions().dtype(radial.dtype()).device(radial.device()));

  torch::Tensor gradX = torch::empty_like(
      X, torch::TensorOptions().dtype(X.dtype()).device(X.device()));

  torch::Tensor gradY = torch::empty_like(
      Y, torch::TensorOptions().dtype(Y.dtype()).device(Y.device()));

  // torch::Tensor sorted_sender_idx =
  // torch::argsort(sender_list).to(torch::kInt32);

  // torch::Tensor sorted_sender_idx = torch::empty_like(sender_list);
  // torch::Tensor first_occurences_node =
  // calculate_first_occurences_gpu_with_sort(sender_list, X.size(0), 128,
  // sorted_sender_idx);

  AT_DISPATCH_FLOATING_TYPES(
      X.type(), "backward_gpu", ([&] {
        dim3 blockDim(NWARPS_PER_BLOCK * WARP_SIZE, 1, 1);
        dim3 gridDim(nnodes, 1);

        void *sptr = nullptr;
        unsigned int space = 0;

        shared_array<scalar_t>(16 * X.size(1), sptr, &space);
        shared_array<scalar_t>(2 * NWARPS_PER_BLOCK * 16, sptr,
                               &space); // buffer_Y, buffer_dY

        void *sptr_node = nullptr;
        unsigned int space_node = 0;

        shared_array<scalar_t>(NWARPS_PER_BLOCK * WARP_SIZE, sptr_node,
                               &space_node);
        shared_array<int32_t>(512 * 2, sptr_node, &space_node);
        // shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr_node,
        // &space_node); // buffer_Y, buffer_dY

        if (nfeatures == 96) {
          backward_edge_inv_tp_kernel<scalar_t, 4,
                                      3><<<gridDim, blockDim, space>>>(
              X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              grad_in
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              gradRadial
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

          backward_node_inv_tp_kernel<scalar_t, 4,
                                      3><<<gridDim, blockDim, space_node>>>(
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              grad_in
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              node_edge_index
                  .packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>(),
              gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());

        } else if (nfeatures == 64) {
          backward_edge_inv_tp_kernel<scalar_t, 4,
                                      2><<<gridDim, blockDim, space>>>(
              X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              grad_in
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              gradRadial
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

          backward_node_inv_tp_kernel<scalar_t, 4,
                                      2><<<gridDim, blockDim, space_node>>>(
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              grad_in
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              node_edge_index
                  .packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>(),
              gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
        } else if (nfeatures == 32) {
          backward_edge_inv_tp_kernel<scalar_t, 4,
                                      1><<<gridDim, blockDim, space>>>(
              X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              grad_in
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              gradRadial
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

          backward_node_inv_tp_kernel<scalar_t, 4,
                                      1><<<gridDim, blockDim, space_node>>>(
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              grad_in
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              node_edge_index
                  .packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>(),
              gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
        } else {
          backward_edge_inv_tp_kernel<scalar_t, 4,
                                      4><<<gridDim, blockDim, space>>>(
              X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              grad_in
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              gradRadial
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

          backward_node_inv_tp_kernel<scalar_t, 4,
                                      4><<<gridDim, blockDim, space_node>>>(
              Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
              radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              grad_in
                  .packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
              sender_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              receiver_list
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              first_occurences
                  .packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
              node_edge_index
                  .packed_accessor64<int32_t, 2, torch::RestrictPtrTraits>(),
              gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
        }
      }));

  return {gradX, gradY, gradRadial};
}