
#include <hip/hip_runtime.h>
#define FULL_MASK 0xffffffff
#define WARP_SIZE 32
#define NWARPS_PER_BLOCK 4

/*
This code has been temporarily transplanted from sphericart to add in some
multipliers directly. Code will be modified
to be able to revert back to sphericart implementaton. In the meantime,\
Please **CITE** sphericart if this code is used in any of your work.

https://github.com/lab-cosmo/sphericart

@article{sphericart,
    title={Fast evaluation of spherical harmonics with sphericart},
    author={Bigi, Filippo and Fraux, Guillaume and Browning, Nicholas J. and
Ceriotti, Michele}, journal={J. Chem. Phys.}, year={2023}, number={159},
    pages={064802},
}
*/

template <typename scalar_t>
__global__ void spherical_harmonics_kernel_ptr(

    const scalar_t *__restrict__ xyz, scalar_t *__restrict__ sph,
    scalar_t *__restrict__ sph_deriv, const int nsamples, const bool normalize,
    const bool requires_grad) {

  extern __shared__ char buffer[];

  int laneID = threadIdx.x % WARP_SIZE;
  int warpID = threadIdx.x / WARP_SIZE;

  void *sptr = buffer;
  unsigned int space = 0;

  const scalar_t sqrt_4pi = 3.5449077018110318;

  scalar_t *buffer_xyz = shared_array<scalar_t>(blockDim.x * 3, sptr, &space);
  scalar_t *buffer_sph = shared_array<scalar_t>(blockDim.x * 16, sptr, &space);

  scalar_t *buffer_sph_deriv_x;
  scalar_t *buffer_sph_deriv_y;
  scalar_t *buffer_sph_deriv_z;

  if (requires_grad) {
    buffer_sph_deriv_x = shared_array<scalar_t>(blockDim.x * 16, sptr, &space);
    buffer_sph_deriv_y = shared_array<scalar_t>(blockDim.x * 16, sptr, &space);
    buffer_sph_deriv_z = shared_array<scalar_t>(blockDim.x * 16, sptr, &space);
  }

  int edge_start = blockIdx.x * blockDim.x;

  for (int i = 0; i < 3; i++) {
    buffer_xyz[i * blockDim.x + threadIdx.x] =
        (edge_start + threadIdx.x < nsamples)
            ? xyz[(edge_start + threadIdx.x) * 3 + i]
            : 0.0;
  }

  __syncthreads();

  // MACE ordering x[:, [2, 0, 1]]
  scalar_t x = buffer_xyz[2 * blockDim.x + threadIdx.x];
  scalar_t y = buffer_xyz[0 * blockDim.x + threadIdx.x];
  scalar_t z = buffer_xyz[1 * blockDim.x + threadIdx.x];

  __syncthreads();

  scalar_t x2 = x * x;
  scalar_t y2 = y * y;
  scalar_t z2 = z * z;

  scalar_t ir = 0.0;

  if (normalize) {
    scalar_t ir2 = 1.0 / (x2 + y2 + z2);
    ir = sqrt(ir2);
    x *= ir;
    y *= ir;
    z *= ir;
    x2 *= ir2;
    y2 *= ir2;
    z2 *= ir2;
  }

  buffer_sph[0 * blockDim.x + threadIdx.x] = 0.282094791773878;

  buffer_sph[1 * blockDim.x + threadIdx.x] = 0.48860251190292 * y;
  buffer_sph[2 * blockDim.x + threadIdx.x] = 0.48860251190292 * z;
  buffer_sph[3 * blockDim.x + threadIdx.x] = 0.48860251190292 * x;

  auto tmp = 2.23606797749979 * x;

  buffer_sph[4 * blockDim.x + threadIdx.x] =
      tmp * buffer_sph[1 * blockDim.x + threadIdx.x];
  buffer_sph[5 * blockDim.x + threadIdx.x] =
      2.23606797749979 * z * buffer_sph[1 * blockDim.x + threadIdx.x];
  buffer_sph[6 * blockDim.x + threadIdx.x] =
      -0.315391565252520 * (x2 + y2 - 2 * z2);
  buffer_sph[7 * blockDim.x + threadIdx.x] =
      tmp * buffer_sph[2 * blockDim.x + threadIdx.x];
  buffer_sph[8 * blockDim.x + threadIdx.x] = 0.54627421529604 * (x2 - y2);

  buffer_sph[9 * blockDim.x + threadIdx.x] =
      -0.59004358992664 * y * (y2 - 3 * x2);
  buffer_sph[10 * blockDim.x + threadIdx.x] =
      2.64575131106459 * z * buffer_sph[4 * blockDim.x + threadIdx.x];
  tmp = -0.457045799464466 * (x2 + y2 - 4 * z2);
  buffer_sph[11 * blockDim.x + threadIdx.x] = y * tmp;
  buffer_sph[12 * blockDim.x + threadIdx.x] =
      -1.49270533036046 * z *
      (z2 - 2.37799637856361 * buffer_sph[6 * blockDim.x + threadIdx.x]);
  buffer_sph[13 * blockDim.x + threadIdx.x] = x * tmp;
  buffer_sph[14 * blockDim.x + threadIdx.x] = 1.44530572132028 * z * (x2 - y2);
  buffer_sph[15 * blockDim.x + threadIdx.x] =
      0.59004358992664 * x * (x2 - 3 * y2);

  __syncthreads();

  for (int i = warpID; i < 16; i += NWARPS_PER_BLOCK) {
    for (int j = laneID; j < blockDim.x; j += WARP_SIZE) {

      if (edge_start + j < nsamples) {
        sph[i * nsamples + edge_start + j] =
            sqrt_4pi * buffer_sph[i * blockDim.x + j];
      }
    }
  }

  if (requires_grad) {
    // dx components first...
    // l = 0
    buffer_sph_deriv_x[0 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_y[0 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_z[0 * blockDim.x + threadIdx.x] = 0.0;
    // l = 1
    buffer_sph_deriv_x[1 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_x[2 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_x[3 * blockDim.x + threadIdx.x] = 0.48860251190292;

    buffer_sph_deriv_y[1 * blockDim.x + threadIdx.x] = 0.48860251190292;
    buffer_sph_deriv_y[2 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_y[3 * blockDim.x + threadIdx.x] = 0.0;

    buffer_sph_deriv_z[1 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_z[2 * blockDim.x + threadIdx.x] = 0.48860251190292;
    buffer_sph_deriv_z[3 * blockDim.x + threadIdx.x] = 0.0;

    // l = 2
    buffer_sph_deriv_x[4 * blockDim.x + threadIdx.x] =
        2.23606797749979 * buffer_sph[1 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_x[5 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_x[6 * blockDim.x + threadIdx.x] =
        -1.29099444873581 * buffer_sph[3 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_x[7 * blockDim.x + threadIdx.x] =
        2.23606797749979 * buffer_sph[2 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_x[8 * blockDim.x + threadIdx.x] =
        2.23606797749979 * buffer_sph[3 * blockDim.x + threadIdx.x];

    buffer_sph_deriv_y[4 * blockDim.x + threadIdx.x] =
        -1.73205080756888 * buffer_sph_deriv_x[6 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_y[5 * blockDim.x + threadIdx.x] =
        buffer_sph_deriv_x[7 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_y[6 * blockDim.x + threadIdx.x] =
        -0.577350269189626 * buffer_sph_deriv_x[4 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_y[7 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_y[8 * blockDim.x + threadIdx.x] =
        -buffer_sph_deriv_x[4 * blockDim.x + threadIdx.x];

    buffer_sph_deriv_z[4 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_z[5 * blockDim.x + threadIdx.x] =
        buffer_sph_deriv_x[4 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_z[6 * blockDim.x + threadIdx.x] =
        1.15470053837925 * buffer_sph_deriv_x[7 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_z[7 * blockDim.x + threadIdx.x] =
        buffer_sph_deriv_y[4 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_z[8 * blockDim.x + threadIdx.x] = 0.0;

    // l = 3
    buffer_sph_deriv_x[9 * blockDim.x + threadIdx.x] =
        3.24037034920393 * buffer_sph[4 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_x[10 * blockDim.x + threadIdx.x] =
        2.64575131106459 * buffer_sph[5 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_x[11 * blockDim.x + threadIdx.x] =
        -0.83666002653408 * buffer_sph[4 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_x[12 * blockDim.x + threadIdx.x] =
        -2.04939015319192 * buffer_sph[7 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_x[13 * blockDim.x + threadIdx.x] =
        0.91409159892893 *
        (y2 - z2 + 4.75599275712721 * buffer_sph[6 * blockDim.x + threadIdx.x]);
    buffer_sph_deriv_x[14 * blockDim.x + threadIdx.x] =
        2.64575131106459 * buffer_sph[7 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_x[15 * blockDim.x + threadIdx.x] =
        3.24037034920393 * buffer_sph[8 * blockDim.x + threadIdx.x];

    buffer_sph_deriv_y[9 * blockDim.x + threadIdx.x] =
        buffer_sph_deriv_x[15 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_y[10 * blockDim.x + threadIdx.x] =
        buffer_sph_deriv_x[14 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_y[11 * blockDim.x + threadIdx.x] =
        -0.91409159892893 *
        (y2 - z2 - 1.58533091904240 * buffer_sph[6 * blockDim.x + threadIdx.x]);
    buffer_sph_deriv_y[12 * blockDim.x + threadIdx.x] =
        -2.04939015319192 * buffer_sph[5 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_y[13 * blockDim.x + threadIdx.x] =
        -0.83666002653408 * buffer_sph[4 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_y[14 * blockDim.x + threadIdx.x] =
        -buffer_sph_deriv_x[10 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_y[15 * blockDim.x + threadIdx.x] =
        -buffer_sph_deriv_x[9 * blockDim.x + threadIdx.x];

    buffer_sph_deriv_z[9 * blockDim.x + threadIdx.x] = 0.0;
    buffer_sph_deriv_z[10 * blockDim.x + threadIdx.x] =
        2.64575131106459 * buffer_sph[4 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_z[11 * blockDim.x + threadIdx.x] =
        3.34664010613630 * buffer_sph[5 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_z[12 * blockDim.x + threadIdx.x] =
        3.54964786985977 * buffer_sph[6 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_z[13 * blockDim.x + threadIdx.x] =
        3.34664010613630 * buffer_sph[7 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_z[14 * blockDim.x + threadIdx.x] =
        2.64575131106459 * buffer_sph[8 * blockDim.x + threadIdx.x];
    buffer_sph_deriv_z[15 * blockDim.x + threadIdx.x] = 0.0;

    __syncthreads();

    for (int j = laneID; j < blockDim.x; j += WARP_SIZE) {

      if (edge_start + j < nsamples) {

        for (int i = warpID; i < 16; i += NWARPS_PER_BLOCK) {

          // MACE ordering x[:, [2, 0, 1]]

          scalar_t tmp_dx = buffer_sph_deriv_x[i * blockDim.x + j];
          scalar_t tmp_dy = buffer_sph_deriv_y[i * blockDim.x + j];
          scalar_t tmp_dz = buffer_sph_deriv_z[i * blockDim.x + j];

          // corrects derivatives for normalization
          if (normalize) {

            scalar_t x = buffer_xyz[2 * blockDim.x + j];
            scalar_t y = buffer_xyz[0 * blockDim.x + j];
            scalar_t z = buffer_xyz[1 * blockDim.x + j];

            scalar_t x2 = x * x;
            scalar_t y2 = y * y;
            scalar_t z2 = z * z;

            scalar_t ir2 = 1.0 / (x2 + y2 + z2);

            scalar_t ir = sqrt(ir2);
            x *= ir;
            y *= ir;
            z *= ir;

            scalar_t tmp_n = (tmp_dx * x + tmp_dy * y + tmp_dz * z);

            scalar_t new_tmp_dx = (tmp_dx - x * tmp_n) * ir;
            scalar_t new_tmp_dy = (tmp_dy - y * tmp_n) * ir;
            scalar_t new_tmp_dz = (tmp_dz - z * tmp_n) * ir;

            sph_deriv[i * 3 * nsamples + 0 * nsamples + edge_start + j] =
                sqrt_4pi * new_tmp_dx;
            sph_deriv[i * 3 * nsamples + 1 * nsamples + edge_start + j] =
                sqrt_4pi * new_tmp_dy;
            sph_deriv[i * 3 * nsamples + 2 * nsamples + edge_start + j] =
                sqrt_4pi * new_tmp_dz;

          } else {
            sph_deriv[i * 3 * nsamples + 0 * nsamples + edge_start + j] =
                sqrt_4pi * tmp_dx;
            sph_deriv[i * 3 * nsamples + 1 * nsamples + edge_start + j] =
                sqrt_4pi * tmp_dy;
            sph_deriv[i * 3 * nsamples + 2 * nsamples + edge_start + j] =
                sqrt_4pi * tmp_dz;
          }
        }
      }
    }
  }
}

template <typename scalar_t>
__global__ void spherical_harmonics_backward_kernel_ptr(
    const scalar_t *__restrict__ sph_deriv,
    const scalar_t *__restrict__ grad_output, const int nsamples,
    scalar_t *__restrict__ xyz_grad) {

  extern __shared__ char buffer[];

  int laneID = threadIdx.x % 16;
  int warpID = threadIdx.x / 16;

  void *sptr = buffer;
  unsigned int space = 0;

  scalar_t *buffer_sum = shared_array<scalar_t>(blockDim.x * 3, sptr, &space);

  // sph_deriv: 16, 3, nsamples;
  // grad: 16, nsamples
  //  xyz: nsamples, 3

  int k_to_idx[3] = {2, 0, 1};

  int edge_start = blockIdx.x * blockDim.x;

  for (int j = warpID; j < blockDim.x; j += 8) {

    scalar_t g = (edge_start + j < nsamples)
                     ? grad_output[laneID * nsamples + edge_start + j]
                     : 0.0;

    for (int k = 0; k < 3; k++) {

      scalar_t sph =
          (edge_start + j < nsamples)
              ? sph_deriv[laneID * 3 * nsamples + k * nsamples + edge_start + j]
              : 0.0;

      scalar_t prod = sph * g;

      // reduce across the sub-warp
      for (int offset = 8; offset > 0; offset /= 2) {
        prod += __shfl_down_sync(FULL_MASK, prod, offset);
      }

      if (laneID == 0) {
        if (edge_start + j < nsamples)
          xyz_grad[(edge_start + j) * 3 + k_to_idx[k]] = prod;
      }
    }
  }
}