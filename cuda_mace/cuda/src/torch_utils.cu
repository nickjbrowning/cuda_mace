#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "torch_utils.cuh"
#include "cuda_utils.cuh"

#define NELEMENTS_PER_BLOCK 512

using namespace std;

__global__ void calculate_first_occurences_kernel(
    const int32_t *__restrict__ receiver_list,
    const int32_t nelements_input,
    const int32_t nelements_output,
    const int32_t *__restrict__ sort_idx,
    bool use_sort,
    int32_t *first_occurences_start,
    int32_t *first_occurences_end)
{

  extern __shared__ char buffer[];
  size_t offset = 0;
  int32_t *smem = reinterpret_cast<int32_t *>(buffer + offset);

  int32_t block_start = blockIdx.x * NELEMENTS_PER_BLOCK;

  // load all elements of senderlist needed by block into shared memory
  for (int32_t i = threadIdx.x; i < NELEMENTS_PER_BLOCK + 1; i += blockDim.x)
  {
    int32_t idx = block_start + i;

    if (idx < nelements_input)
    {
      if (use_sort)
      {
        smem[i] = receiver_list[sort_idx[idx]];
      }
      else
      {
        smem[i] = receiver_list[idx];
      }
    }
  }

  __syncthreads();

  // deal with even boundaries
  for (int32_t i = 2 * threadIdx.x; i < NELEMENTS_PER_BLOCK; i += 2 * blockDim.x)
  {
    int32_t idx = block_start + i;

    if (idx + 1 < nelements_input)
    {
      int32_t loc1 = smem[i];
      int32_t loc2 = smem[i + 1];

      if (loc1 != loc2)
      {
        first_occurences_end[loc1] = idx + 1;
        first_occurences_start[loc2] = idx + 1;
      }
    }
  }

  // deal with odd boundaries
  for (int32_t i = 2 * threadIdx.x + 1; i < NELEMENTS_PER_BLOCK + 1; i += 2 * blockDim.x)
  {
    int32_t idx = block_start + i;

    if (idx + 1 < nelements_input)
    {
      int32_t loc1 = smem[i];
      int32_t loc2 = smem[i + 1];

      if (loc1 != loc2)
      {
        first_occurences_end[loc1] = idx + 1;
        first_occurences_start[loc2] = idx + 1;
      }
    }
  }

  // deal with 0th and last element specifically, so we dont need to use torch::zeros
  if (blockIdx.x == 0 && threadIdx.x == 0)
  {
    first_occurences_start[receiver_list[0]] = 0;
    first_occurences_end[receiver_list[nelements_input - 1]] = nelements_input;
  }
}

torch::Tensor calculate_first_occurences_gpu(torch::Tensor receiver_list,
                                             int64_t nnodes, int64_t nthreadx)
{
  torch::Tensor first_occurences =
      torch::empty(2 * nnodes, torch::TensorOptions()
                               .dtype(receiver_list.dtype())
                               .device(receiver_list.device()));

  int32_t nbx =
      find_integer_divisor(receiver_list.size(0), NELEMENTS_PER_BLOCK);

  dim3 block_dim(nbx);

  dim3 grid_dim(nthreadx, 1, 1);

  size_t total_buff_size = 0;

  total_buff_size += (NELEMENTS_PER_BLOCK + 1) * sizeof(int32_t);

  calculate_first_occurences_kernel<<<block_dim, grid_dim, total_buff_size>>>(
      receiver_list.data_ptr<int32_t>(),
      receiver_list.size(0),
      nnodes,
      nullptr, 
      false, 
      first_occurences.data_ptr<int32_t>(),
      first_occurences.data_ptr<int32_t>() + nnodes);

  return first_occurences;
}