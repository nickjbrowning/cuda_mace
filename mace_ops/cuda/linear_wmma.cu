#include "hip/hip_runtime.h"
#include <mma.h>
#include <hip/hip_runtime.h>
#include <torch/script.h>

#include <stdio.h>
#include <stdlib.h>
#include <cuda/barrier>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_fp16.h>
#include <cuda_pipeline_primitives.h>

using namespace nvcuda;
using namespace std;
using namespace torch::autograd;

#define WARP_SIZE 32

// MMA matrix tile dimensions.
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 8

__host__ __device__ int32_t find_integer_divisor(int32_t x, int32_t bdim)
{
    return (x + bdim - 1) / bdim;
}

template <class T>
__host__ __device__ T *shared_array(std::size_t n_elements, void *&ptr,
                                    std::size_t *space = nullptr) noexcept
{
    const std::uintptr_t inptr = reinterpret_cast<uintptr_t>(ptr);
    const std::uintptr_t end = inptr + n_elements * sizeof(T);
    if (space)
        *space += static_cast<std::size_t>(end - inptr);
    ptr = reinterpret_cast<void *>(end);
    return reinterpret_cast<T *>(inptr);
}

template <class T>
__host__ __device__ T *align_array(std::size_t n_elements, void *&ptr, const std::size_t alignment,
                                   std::size_t *space = nullptr) noexcept
{
    // const std::size_t alignment = alignof(T);
    const std::uintptr_t intptr = reinterpret_cast<uintptr_t>(ptr);
    const std::uintptr_t aligned = (intptr + alignment - 1) & -alignment;
    const std::uintptr_t end = aligned + n_elements * sizeof(T);
    if (space)
        *space += static_cast<std::size_t>(end - intptr);
    ptr = reinterpret_cast<void *>(end);
    return reinterpret_cast<T *>(aligned);
}

__global__ void matmul_wmma_pipeline_kernel(float *X, float *W, float *OUT, const int NNODES, const int M_TOTAL, const int N_TOTAL, const int K_TOTAL)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    float *buffer_X = shared_array<float>(blockDim.x * M_TOTAL, sptr, &space);
    float *buffer_W = shared_array<float>(blockDim.x * N_TOTAL, sptr, &space);

    float *X_i = X + blockIdx.x * M_TOTAL * K_TOTAL;

    __syncthreads();

    // wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> ab_frag;

    wmma::fill_fragment(ab_frag, 0.0f);

    int aRow = 0;
    int bCol = (blockIdx.y * blockDim.y + threadIdx.y) * WMMA_N;

    // do matmuls over 32 chunk of K_TOTAL
    for (int kchunk = 0; kchunk < find_integer_divisor(K_TOTAL, blockDim.x); kchunk++)
    {

        // pre-fetch 16x32 chunk of X, and 32x128 chunk of W
        for (int j = threadIdx.y; j < M_TOTAL; j += blockDim.y)
        {
            __pipeline_memcpy_async(buffer_X + j * blockDim.x + threadIdx.x, X_i + j * K_TOTAL + threadIdx.x, sizeof(float));
        }

        for (int j = threadIdx.y; j < blockDim.x; j += blockDim.y)
        {
            for (int i = threadIdx.x; i < N_TOTAL; i += blockDim.x)
            {
                __pipeline_memcpy_async(buffer_W + j * N_TOTAL + i, W + j * N_TOTAL + i, sizeof(float));
            }
        }

        __pipeline_commit();

        __pipeline_wait_prior(0);

        // do the computation over K
        for (int i = 0; i < blockDim.x; i += WMMA_K)
        {
            wmma::load_matrix_sync(a_frag, buffer_X + aRow * blockDim.x + i, blockDim.x);
            wmma::load_matrix_sync(b_frag, buffer_W + bCol + i * N_TOTAL, N_TOTAL);

            // Perform the matrix multiplication
            wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
        }
    }

    wmma::store_matrix_sync(OUT + blockIdx.x * M_TOTAL * N_TOTAL + bCol + aRow * N_TOTAL, ab_frag, N_TOTAL, wmma::mem_row_major);
}

__global__ void matmul_wmma_kernel(float *X, float *W, float *OUT, const int NNODES, const int M_TOTAL, const int N_TOTAL, const int K_TOTAL)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    float *buffer_X = shared_array<float>(K_TOTAL * M_TOTAL, sptr, &space);

    float *X_i = X + blockIdx.x * M_TOTAL * K_TOTAL;

    for (int j = threadIdx.y; j < M_TOTAL; j += blockDim.y)
    {
        for (int k = threadIdx.x; k < K_TOTAL; k += blockDim.x)
        {
            buffer_X[k * M_TOTAL + j] = X_i[j * K_TOTAL + k];
        }
    }

    __syncthreads();

    // wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> ab_frag;

    wmma::fill_fragment(ab_frag, 0.0f);

    int aRow = 0;
    int bCol = (blockIdx.y * blockDim.y + threadIdx.y) * WMMA_N;

    for (int i = 0; i < K_TOTAL; i += WMMA_K)
    {
        wmma::load_matrix_sync(a_frag, buffer_X + i * M_TOTAL + aRow, M_TOTAL);
        wmma::load_matrix_sync(b_frag, W + bCol + i * N_TOTAL, N_TOTAL);

        // Perform the matrix multiplication
        wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
    }

    wmma::store_matrix_sync(OUT + blockIdx.x * M_TOTAL * N_TOTAL + bCol + aRow * N_TOTAL, ab_frag, N_TOTAL, wmma::mem_row_major);
}

__global__ void matmul_wmma_with_correction_kernel(float *X, float *W, float *OUT, const int NNODES, const int M_TOTAL, const int N_TOTAL, const int K_TOTAL)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    float *buffer_X = shared_array<float>(K_TOTAL * M_TOTAL, sptr, &space);
    float *buffer_delta_X = shared_array<float>(K_TOTAL * M_TOTAL, sptr, &space);
    float *buffer_W = shared_array<float>(WMMA_K * N_TOTAL, sptr, &space); // could avoid this if we stored W and dW in global memory...
    float *buffer_delta_W = shared_array<float>(WMMA_K * N_TOTAL, sptr, &space);

    for (int i = threadIdx.y; i < M_TOTAL; i += blockDim.y)
    {
        for (int j = threadIdx.x; j < K_TOTAL; j += blockDim.x)
        {
            float x = X[blockIdx.x * M_TOTAL * K_TOTAL + i * K_TOTAL + j];

            float xtf32 = wmma::__float_to_tf32(x);

            buffer_X[j * M_TOTAL + i] = xtf32;
            buffer_delta_X[j * M_TOTAL + i] = wmma::__float_to_tf32(x - xtf32);
        }
    }

    __syncthreads();

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::col_major> a_frag, delta_a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, wmma::precision::tf32, wmma::row_major> b_frag, delta_b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> ab_frag;

    wmma::fill_fragment(ab_frag, 0.0f);

    int aRow = 0;
    int bCol = (blockIdx.y * blockDim.y + threadIdx.y) * WMMA_N;

    for (int i = 0; i < K_TOTAL; i += WMMA_K)
    {

        for (int j = threadIdx.y; j < WMMA_K; j += blockDim.y)
        {
            for (int k = threadIdx.x; k < N_TOTAL; k += blockDim.x)
            {
                float w = W[(i + j) * N_TOTAL + k];
                float wtf32 = wmma::__float_to_tf32(w);

                buffer_W[j * N_TOTAL + k] = wtf32;
                buffer_delta_W[j * N_TOTAL + k] = wmma::__float_to_tf32(w - wtf32);
            }
        }

        // void cuda::memcpy_async(void* destination, void const* source, Shape size, cuda::pipeline<Scope>& pipeline);

        //  Now need to compute C_{32} = A_{16}B_{16} + \Delta A_{16} B_{16} + A_{16}\Delta B_{16} + \Delta A_{16}\Delta B_{16}
        //\Delta A_{16}\Delta B_{16} is very small relative correction so can be ignored.
        __syncthreads();

        wmma::load_matrix_sync(a_frag, buffer_X + i * M_TOTAL + aRow, M_TOTAL);
        wmma::load_matrix_sync(delta_a_frag, buffer_delta_X + i * M_TOTAL + aRow, M_TOTAL);

        // wmma::load_matrix_sync(b_frag, buffer_W + bCol + bRow * ldb, ldb);
        // wmma::load_matrix_sync(delta_b_frag, buffer_W + bCol + bRow * ldb, ldb);

        wmma::load_matrix_sync(b_frag, buffer_W + bCol, N_TOTAL);
        wmma::load_matrix_sync(delta_b_frag, buffer_delta_W + bCol, N_TOTAL);

        // Perform the matrix multiplication
        wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
        wmma::mma_sync(ab_frag, delta_a_frag, b_frag, ab_frag);
        wmma::mma_sync(ab_frag, a_frag, delta_b_frag, ab_frag);
        // wmma::mma_sync(ab_frag, delta_a_frag, delta_b_frag, ab_frag);
    }

    __syncthreads();

    wmma::store_matrix_sync(OUT + blockIdx.x * M_TOTAL * N_TOTAL + bCol + aRow * N_TOTAL, ab_frag, N_TOTAL, wmma::mem_row_major);
}

torch::Tensor matmul_wmma(torch::Tensor X, torch::Tensor W, bool error_corrected)
{
    const int NNODES = X.size(0);
    const int M = X.size(1);
    const int N = W.size(1);
    const int K = W.size(0);

    TORCH_CHECK(X.device().is_cuda(), "X must be a CUDA tensor");
    TORCH_CHECK(W.device().is_cuda(), "W must be a CUDA tensor");

    TORCH_CHECK(M == 16, "X dim=1 must have dimension 16 [(lmax +1)**2]");
    TORCH_CHECK(N % 16 == 0, "W dim=2 must be a multiple of 16");
    TORCH_CHECK(K % 16 == 0, "X dim=2 must be a multiple of 16");

    torch::Tensor output = torch::empty({NNODES, M, N},
                                        torch::TensorOptions()
                                            .dtype(X.dtype())
                                            .device(X.device()));

    dim3 gridDim, blockDim;

    blockDim.x = WARP_SIZE;
    blockDim.y = 8;

    gridDim.x = NNODES;
    gridDim.y = find_integer_divisor(N, blockDim.y * WMMA_N);

    size_t shared_size = 0;
    void *sptr = nullptr;

    assert(((unsigned long long)X.data_ptr<float>()) % 128 == 0);
    assert(((unsigned long long)W.data_ptr<float>()) % 128 == 0);
    assert(((unsigned long long)output.data_ptr<float>()) % 128 == 0);

    if (!error_corrected)
    {
        shared_array<float>(K * M, sptr, &shared_size);
        // shared_array<float>(WARP_SIZE * M, sptr, &shared_size);
        // shared_array<float>(WARP_SIZE * N, sptr, &shared_size);

        matmul_wmma_kernel<<<gridDim, blockDim, shared_size>>>(X.data_ptr<float>(), W.data_ptr<float>(), output.data_ptr<float>(),
                                                               NNODES, M, N, K);
    }
    else
    {
        shared_array<float>(K * M, sptr, &shared_size);
        shared_array<float>(K * M, sptr, &shared_size);
        shared_array<float>(WMMA_K * N, sptr, &shared_size);
        shared_array<float>(WMMA_K * N, sptr, &shared_size);
        matmul_wmma_with_correction_kernel<<<gridDim, blockDim, shared_size>>>(X.data_ptr<float>(), W.data_ptr<float>(), output.data_ptr<float>(),
                                                                               NNODES, M, N, K);
    }

    // hipDeviceSynchronize();

    // torch::Tensor output = torch::from_blob(output_ptr, {NNODES, M, N}, deleter, torch::TensorOptions().dtype(X.dtype()).device(X.device()));

    return output;
}

class MatmulAutograd : public Function<MatmulAutograd>
{
public:
    static torch::Tensor forward(
        AutogradContext *ctx,
        torch::Tensor X,
        torch::Tensor W,
        torch::Tensor W_transposed,
        bool error_corrected)
    {
        if (X.requires_grad())
        {
            ctx->save_for_backward({W_transposed});
        }

        ctx->saved_data["error_corrected"] = error_corrected;

        return matmul_wmma(X, W, error_corrected);
    }

    static variable_list backward(AutogradContext *ctx, variable_list grad_outputs)
    {
        auto saved_variables = ctx->get_saved_variables();

        auto W_T = saved_variables[0];
        bool error_corrected = ctx->saved_data["error_corrected"].toBool();

        torch::Tensor dX = matmul_wmma(grad_outputs[0].contiguous(), W_T, error_corrected);

        torch::Tensor undef;

        return {dX, undef, undef, undef};
    }
};

torch::Tensor matmul(
    torch::Tensor X,
    torch::Tensor W,
    torch::Tensor W_T,
    bool error_corrected)
{
    return MatmulAutograd::apply(X, W, W_T, error_corrected);
}

__global__ void linear_wmma_kernel(
    const float *__restrict__ X,
    const float *__restrict__ W,
    float *OUT,
    const int *__restrict__ l_start,
    const int *__restrict__ l_end,
    const float *__restrict__ path_weights,
    const int ninstructions,
    const int NNODES,
    const int M_TOTAL,
    const int N_TOTAL,
    const int K_TOTAL)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    float *buffer_X = shared_array<float>(K_TOTAL * M_TOTAL, sptr, &space);
    float *buffer_tmp_output = shared_array<float>(M_TOTAL * N_TOTAL, sptr, &space);

    for (int i = threadIdx.y; i < M_TOTAL; i += blockDim.y)
    {
        for (int j = threadIdx.x; j < K_TOTAL; j += blockDim.x)
        {
            buffer_X[j * M_TOTAL + i] = X[blockIdx.x * M_TOTAL * K_TOTAL + i * K_TOTAL + j];
        }
    }

    __syncthreads();

    int a_row = 0;
    int b_col = (blockIdx.y * blockDim.y + threadIdx.y) * WMMA_N;

    for (int instruction = 0; instruction < ninstructions; instruction++)
    {
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, 8, wmma::precision::tf32, wmma::col_major> a_frag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, 8, wmma::precision::tf32, wmma::row_major> b_frag;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, 8, float> ab_frag;

        int lstart = l_start[instruction];
        int lend = l_end[instruction];
        float pathw = path_weights[instruction];

        wmma::fill_fragment(ab_frag, 0.0f);

        for (int k = 0; k < K_TOTAL; k += 8)
        {

            wmma::load_matrix_sync(a_frag, buffer_X + k * M_TOTAL + a_row, M_TOTAL);
            wmma::load_matrix_sync(b_frag, W + (instruction * K_TOTAL * N_TOTAL) + b_col + k * N_TOTAL, N_TOTAL);

            // Perform the matrix multiplication
            wmma::mma_sync(ab_frag, a_frag, b_frag, ab_frag);
        }

        // apply path weight
        for (int i = 0; i < ab_frag.num_elements; i++)
        {
            ab_frag.x[i] = ab_frag.x[i] * pathw;
        }

        wmma::store_matrix_sync(buffer_tmp_output + b_col + a_row * N_TOTAL, ab_frag, N_TOTAL, wmma::mem_row_major);

        // wait for output to be fully populated...
        __syncthreads();

        // write out the part of the matmul that we need.
        for (int lm = lstart + threadIdx.y; lm < lend; lm += blockDim.y)
        {
            for (int channel = threadIdx.x; channel < N_TOTAL; channel += blockDim.x)
            {
                OUT[blockIdx.x * M_TOTAL * N_TOTAL + lm * N_TOTAL + channel] = buffer_tmp_output[lm * N_TOTAL + channel];
            }
        }
    }
}

torch::Tensor linear_(
    torch::Tensor X,
    torch::Tensor W,
    torch::Tensor l_start,
    torch::Tensor l_end,
    torch::Tensor path_weights,
    bool print_debug = false)
{

    const int NNODES = X.size(0);
    const int M = X.size(1);
    const int ninstructions = W.size(0);
    const int N = W.size(2);
    const int K = W.size(1);

    TORCH_CHECK(X.device().is_cuda(), "X must be a CUDA tensor");
    TORCH_CHECK(W.device().is_cuda(), "W must be a CUDA tensor");
    TORCH_CHECK(l_start.device().is_cuda(), "l_start must be a CUDA tensor");
    TORCH_CHECK(l_end.device().is_cuda(), "l_end must be a CUDA tensor");

    TORCH_CHECK(l_start.size(0) == l_end.size(0) && l_start.size(0) == W.size(0), "l_start/end must be same size as first dimension of W");

    TORCH_CHECK(M == 16, "X dim=1 must have dimension 16 [(lmax +1)**2]");
    TORCH_CHECK(N % 16 == 0, "W dim=2 must be a multiple of 16");
    TORCH_CHECK(K % 16 == 0, "X dim=2 must be a multiple of 16");

    torch::Tensor output = torch::empty({NNODES, M, N},
                                        torch::TensorOptions()
                                            .dtype(X.dtype())
                                            .device(X.device()));

    dim3 gridDim, blockDim;
    blockDim.x = WARP_SIZE;
    blockDim.y = 8; // 8 * WMMA_N = 64

    gridDim.x = NNODES;
    gridDim.y = find_integer_divisor(N, blockDim.y * WMMA_N);

    // std::cout << "grid dim: " << gridDim.x << " " << gridDim.y << " " << gridDim.z << std::endl;
    // std::cout << "block dim: " << blockDim.x << " " << blockDim.y << std::endl;

    size_t shared_size = 0;
    void *sptr = nullptr;

    shared_array<float>(M * K, sptr, &shared_size); // X
    shared_array<float>(M * N, sptr, &shared_size); // tmp_output

    linear_wmma_kernel<<<gridDim, blockDim, shared_size>>>(X.data_ptr<float>(), W.data_ptr<float>(), output.data_ptr<float>(),
                                                           l_start.data_ptr<int>(),
                                                           l_end.data_ptr<int>(),
                                                           path_weights.data_ptr<float>(),
                                                           ninstructions,
                                                           NNODES, M, N, K);

    hipDeviceSynchronize();

    return output;
}

class LinearAutograd : public Function<LinearAutograd>
{
public:
    static torch::Tensor forward(
        AutogradContext *ctx,
        torch::Tensor X,
        torch::Tensor W,
        torch::Tensor W_transposed, // needed for backwards pass dL/dX
        torch::Tensor l_start,
        torch::Tensor l_end,
        torch::Tensor path_weights)
    {

        if (X.requires_grad())
        {
            ctx->save_for_backward({W_transposed, l_start, l_end, path_weights});
        }

        torch::Tensor result = linear_(X, W, l_start, l_end, path_weights);

        return result;
    }

    static variable_list backward(AutogradContext *ctx, variable_list grad_outputs)
    {
        auto saved_variables = ctx->get_saved_variables();

        auto W_T = saved_variables[0];
        auto l_start = saved_variables[1];
        auto l_end = saved_variables[2];
        auto path_weights = saved_variables[3];

        torch::Tensor dX = linear_(grad_outputs[0], W_T, l_start, l_end, path_weights);

        torch::Tensor dW;

        // if (W.requires_grad())
        //{
        //  for i  in range(x.shape[0]) : grad_w += torch.matmul(x[i].transpose(-1, -2), grad_output[i])
        // dW = torch::bmm(X.transpose(-1, -2).contiguous(), grad_outputs[0]).sum(0);
        //}

        torch::Tensor undef;

        return {dX, undef, undef, undef, undef, undef};
    }
};

torch::Tensor linear(
    torch::Tensor X,
    torch::Tensor W,
    torch::Tensor W_T,
    torch::Tensor l_start,
    torch::Tensor l_end,
    torch::Tensor path_weights)
{
    return LinearAutograd::apply(X, W, W_T, l_start, l_end, path_weights);
}

TORCH_LIBRARY(linear_wmma, m)
{
    m.def("linear", &linear);
    m.def("matmul", &matmul);
    m.def("matmul_base", &matmul_wmma);
}
