#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include "torch_utils.cuh"
#include "invariant_message_passing_impl.cuh"

#include <torch/script.h>
#include <iostream>

using namespace std;
using namespace torch::indexing;
using namespace torch::autograd;

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)

#define FULL_MASK 0xffffffff

#define WARP_SIZE 32
#define NWARPS_PER_BLOCK 4

template <typename scalar_t, const int TM, const int TN>
__global__ void inv_tp_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> X, // [nnodes nchannels]
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> Y, // [nedges, (L+1)**2]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> radial,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> sender_list,      //
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> receiver_list,    // which index we need to sum a particular edge into -> monotonically increasing.
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> first_occurences, // the indexes in reciever_list which deliniate the set of edges per node.
    torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> output)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    int32_t *buffer_sender = shared_array<int32_t>(512, sptr, &space);

    const uint threadCol = threadIdx.x % WARP_SIZE;
    const uint threadRow = threadIdx.x / WARP_SIZE;

    const uint N = X.size(1);
    const uint edge_start = first_occurences[blockIdx.x];
    const uint edge_end = (blockIdx.x == first_occurences.size(0) - 1) ? receiver_list.size(0) : first_occurences[blockIdx.x + 1];
    const uint node_index = receiver_list[edge_start];

    scalar_t regY[TM] = {0.0};
    scalar_t regX[TN] = {0.0};
    scalar_t regRadial[TM * TN] = {0.0};
    scalar_t regOut[TN * TM] = {0.0};
    scalar_t regC[TN * TM] = {0.0};
    // check if this node has neighbours
    if (edge_end - edge_start == 0)
    {
        return;
    }

    if (edge_end - edge_start > 512 && threadIdx.x == 0)
    {
        printf("WARNING: Cuda Invariant Message Passing only supports at most 512 neighbours, node: %d has %d.\n", node_index, (edge_end - edge_start));
    }

    for (int tid = threadIdx.x; tid < edge_end - edge_start; tid += blockDim.x)
    {
        buffer_sender[tid] = sender_list[edge_start + tid];
    }

    __syncthreads();

    for (int feature = threadCol; feature < N; feature += WARP_SIZE * TN)
    {
        for (int m = threadRow; m < 16; m += NWARPS_PER_BLOCK * TM)
        {

            for (int i = 0; i < TN; i++)
            {
                for (int j = 0; j < TM; j++)
                {
                    regC[i * TM + j] = 0.0;
                    regOut[i * TM + j] = 0.0;
                }
            }

            for (uint edge = edge_start; edge < edge_end; edge++)
            {
                for (int i = 0; i < TN; i++)
                {
                    regX[i] = X[buffer_sender[edge - edge_start]][i * WARP_SIZE + feature];
                }
                for (int i = 0; i < TN; i++)
                {
                    for (int j = 0; j < TM; j++)
                    {
                        int32_t lm_index = sqrt(j * NWARPS_PER_BLOCK + m);

                        regRadial[i * TM + j] = radial[edge][lm_index][i * WARP_SIZE + feature];
                    }
                }

                for (int j = 0; j < TM; j++)
                {
                    regY[j] = Y[edge][j * NWARPS_PER_BLOCK + m];
                }

                for (int i = 0; i < TN; i++)
                {
                    for (int j = 0; j < TM; j++)
                    {
                        scalar_t val = regX[i] * regY[j] * regRadial[i * TM + j];
                        scalar_t val_compensated = val - regC[i * TM + j];
                        scalar_t tmp_new = regOut[i * TM + j] + val_compensated;
                        regC[i * TM + j] = (tmp_new - regOut[i * TM + j]) - val_compensated;
                        regOut[i * TM + j] = tmp_new;
                    }
                }
            }

            for (int i = 0; i < TN; i++)
            {
                for (int j = 0; j < TM; j++)
                {
                    output[node_index][j * NWARPS_PER_BLOCK + m][i * WARP_SIZE + feature] = regOut[i * TM + j];
                }
            }
        }
    }
}

torch::Tensor forward_gpu(
    torch::Tensor X,
    torch::Tensor Y,
    torch::Tensor radial,
    torch::Tensor sender_list,
    torch::Tensor receiver_list,
    torch::Tensor first_occurences,
    const int64_t nnodes)
{

    const uint nedges = Y.size(0);
    const int nspherical_harm = Y.size(1);
    const int nfeatures = X.size(1);

    TORCH_CHECK(nfeatures % WARP_SIZE == 0, "feature dimension must be a multiple of 32");
    TORCH_CHECK(nspherical_harm == 16, "number of edge spherical harmonics must be 16");
    TORCH_CHECK(nfeatures <= 128, "feature dimension cannot be greater than 128");

    torch::Tensor output = torch::empty({nnodes, nspherical_harm, nfeatures},
                                        torch::TensorOptions()
                                            .dtype(X.dtype())
                                            .device(X.device()));

    dim3 gridDim(nnodes);

    dim3 blockDim(NWARPS_PER_BLOCK * WARP_SIZE, 1, 1);

    AT_DISPATCH_FLOATING_TYPES(
        X.type(), "forward_gpu", ([&]
                                  {
            size_t space = 0;
            void *sptr = nullptr;

            shared_array<int32_t>(512, sptr, &space);

            if (nfeatures >= 128)
            {
                inv_tp_kernel<scalar_t, 4, 4><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    output.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
            }
            else if (nfeatures == 96)
            {
                inv_tp_kernel<scalar_t, 4, 3><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    output.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
            }else if (nfeatures == 64)
            {
                inv_tp_kernel<scalar_t, 4, 2><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    output.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
            }else if (nfeatures == 32)
            {
                inv_tp_kernel<scalar_t, 4, 1><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    output.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
            } }

                                  ));

    hipDeviceSynchronize();

    return output;
}

template <typename scalar_t, const int TM, const int TN>
__global__ void backward_edge_inv_tp_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> X,       // [nedges, feat]
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> Y,       // [nedges, m]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> radial,  // [nedges, LMAX, feat]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> grad_in, // [nnodes, m, feat]
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> sender_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> receiver_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> first_occurences,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> gradY,
    torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> grad_radial)
{
    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    scalar_t *buffer_grad_in = shared_array<scalar_t>(16 * X.size(1), sptr, &space);
    scalar_t *buffer_Y = shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr, &space);
    scalar_t *buffer_dY = shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr, &space);

    const uint threadCol = threadIdx.x % WARP_SIZE;
    const uint threadRow = threadIdx.x / WARP_SIZE;

    const uint edge_start = first_occurences[blockIdx.x];
    const uint node_index = receiver_list[edge_start];
    const uint edge_end = (blockIdx.x == first_occurences.size(0) - 1) ? receiver_list.size(0) : first_occurences[blockIdx.x + 1];

    const uint N_start = blockIdx.y * TN * WARP_SIZE;

    scalar_t regX[TN] = {0.0};
    scalar_t regW[4 * TN] = {0.0};

    scalar_t regGradW[4 * TN] = {0.0};

    if (edge_end - edge_start == 0)
    {
        return;
    }

    for (int m = 0; m < 16 / NWARPS_PER_BLOCK; m++)
    {
        for (int n = 0; n < TN; n++)
        {
            // if (N_start + n * WARP_SIZE + threadCol < X.size(1))
            buffer_grad_in[(m * NWARPS_PER_BLOCK + threadRow) * X.size(1) + n * WARP_SIZE + threadCol] = grad_in[node_index][m * NWARPS_PER_BLOCK + threadRow][N_start + n * WARP_SIZE + threadCol];
        }
    }

    __syncthreads();

    int niter = find_integer_divisor(edge_end - edge_start, NWARPS_PER_BLOCK);

    for (uint ni = 0; ni < niter; ni++)
    {
        uint edge = edge_start + ni * NWARPS_PER_BLOCK + threadRow;

        if (edge < edge_end)
        {
            uint sender_id = sender_list[edge];

            if (threadCol < 16)
            {
                buffer_Y[threadCol * NWARPS_PER_BLOCK + threadRow] = Y[edge][threadCol];
                buffer_dY[threadCol * NWARPS_PER_BLOCK + threadRow] = 0.0;
            }

            __syncwarp();

            for (int n = 0; n < TN; n++)
            {

                regX[n] = X[sender_id][N_start + n * WARP_SIZE + threadCol];
            }

            for (int n = 0; n < TN; n++)
            {
                for (int L = 0; L < 4; L++)
                {
                    regGradW[L * TN + n] = 0.0;
                    regW[L * TN + n] = radial[edge][L][N_start + n * WARP_SIZE + threadCol];
                }
            }

            for (int L = 0; L < 4; L++)
            {
                uint mstart = L * L;
                uint mend = (L + 1) * (L + 1);

                for (int m = mstart; m < mend; m++)
                {
                    scalar_t sph = buffer_Y[m * NWARPS_PER_BLOCK + threadRow]; // Y[edge][m];

                    scalar_t dgradY = 0.0;

                    for (int n = 0; n < TN; n++)
                    {
                        //  scalar_t gradin = regGradIn[m * TN + n];
                        scalar_t gradin = buffer_grad_in[m * X.size(1) + n * WARP_SIZE + threadCol];
                        scalar_t w = regW[L * TN + n];

                        regGradW[L * TN + n] += sph * regX[n] * gradin;

                        dgradY += gradin * w * regX[n];
                    }

                    for (int offset = 16; offset > 0; offset /= 2)
                    {
                        dgradY += __shfl_down_sync(FULL_MASK, dgradY, offset, WARP_SIZE);
                    }

                    // threadIdx % WARP_SIZE = 0 dgradY contains the derivative of the output wrt. Y
                    if (threadCol == 0)
                        buffer_dY[m * NWARPS_PER_BLOCK + threadRow] = dgradY;
                }
            }
        }

        __syncthreads();

        if (edge < edge_end)
        {
            if (threadCol < 16)
            {
                gradY[edge][threadCol] = buffer_dY[threadCol * NWARPS_PER_BLOCK + threadRow];
            }

            for (int n = 0; n < TN; n++)
            {
                //  write grad_radial
                for (int L = 0; L < 4; L++)
                {
                    grad_radial[edge][L][N_start + n * WARP_SIZE + threadCol] = regGradW[L * TN + n];
                }
            }
        }
    }
}

template <typename scalar_t, const int TM, const int TN>
__global__ void backward_node_inv_tp_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> Y,
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> radial,
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> grad_in,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> sender_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> receiver_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> sorted_sender_idx,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> first_occurences,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> gradX)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    scalar_t regY[TM] = {0.0};
    scalar_t regRadial[TM * TN] = {0.0};
    scalar_t regGradIn[TM * TN] = {0.0};
    scalar_t regGradX[TN] = {0.0};
    scalar_t regGradXC[TN] = {0.0};

    scalar_t *buffer_out = shared_array<scalar_t>(NWARPS_PER_BLOCK * WARP_SIZE, sptr, &space);
    int32_t *buffer_sorted_sender_idx = shared_array<int32_t>(512, sptr, &space);
    int32_t *buffer_receiver_list = shared_array<int32_t>(512, sptr, &space);

    const uint threadCol = threadIdx.x % WARP_SIZE;
    const uint threadRow = threadIdx.x / WARP_SIZE;

    const uint edge_start = first_occurences[blockIdx.x];
    const uint node_index = sender_list[sorted_sender_idx[edge_start]];
    const uint edge_end = (blockIdx.x == first_occurences.size(0) - 1) ? sender_list.size(0) : first_occurences[blockIdx.x + 1];

    if (edge_end - edge_start == 0)
    {
        return;
    }

    for (int tid = threadIdx.x; tid < edge_end - edge_start; tid += blockDim.x)
    {
        int32_t sorted_id = sorted_sender_idx[edge_start + tid];
        buffer_sorted_sender_idx[tid] = sorted_id;
        buffer_receiver_list[tid] = receiver_list[sorted_id];
    }

    __syncthreads();

    for (int feature = threadCol; feature < gradX.size(1); feature += WARP_SIZE * TN)
    {
        __syncthreads();

        for (int i = 0; i < TN; i++)
        {
            regGradX[i] = 0.0;
            regGradXC[i] = 0.0;
        }

        for (int32_t edge = edge_start; edge < edge_end; edge++)
        {
            int32_t sorted_id = buffer_sorted_sender_idx[edge - edge_start];
            int32_t receiver_id = buffer_receiver_list[edge - edge_start];

            for (int m = threadRow; m < 16; m += NWARPS_PER_BLOCK * TM)
            {
                for (int j = 0; j < TM; j++)
                {
                    regY[j] = Y[sorted_id][j * NWARPS_PER_BLOCK + m];
                }

                for (int j = 0; j < TM; j++)
                {
                    int32_t lm_index = sqrt(j * NWARPS_PER_BLOCK + m);
                    for (int i = 0; i < TN; i++)
                    {
                        regRadial[i * TM + j] = radial[sorted_id][lm_index][i * WARP_SIZE + feature];
                        regGradIn[i * TM + j] = grad_in[receiver_id][j * NWARPS_PER_BLOCK + m][i * WARP_SIZE + feature];
                    }
                }

                for (int i = 0; i < TN; i++)
                {
                    for (int j = 0; j < TM; j++)
                    {
                        scalar_t val = regGradIn[i * TM + j] * regRadial[i * TM + j] * regY[j];
                        scalar_t val_compensated = val - regGradXC[i];
                        scalar_t tmp_new = regGradX[i] + val_compensated;
                        regGradXC[i] = (tmp_new - regGradX[i]) - val_compensated;
                        regGradX[i] = tmp_new;
                    }
                }
            }
        }

        for (int i = 0; i < TN; i++)
        {
            __syncthreads();

            buffer_out[threadRow * WARP_SIZE + threadCol] = regGradX[i];

            __syncthreads();
            /* need to reduce over m here*/
            if (threadRow == 0)
            {
                scalar_t tmp = 0.0;
                for (int j = 0; j < NWARPS_PER_BLOCK; j++)
                {
                    tmp += buffer_out[j * WARP_SIZE + threadCol];
                }

                gradX[node_index][i * WARP_SIZE + feature] = tmp;
            }
        }
    }
}

std::vector<torch::Tensor> backward_gpu(torch::Tensor X,
                                        torch::Tensor Y,
                                        torch::Tensor radial,
                                        torch::Tensor grad_in,
                                        torch::Tensor sender_list,
                                        torch::Tensor receiver_list,
                                        torch::Tensor first_occurences,
                                        const int64_t nnodes)
{
    uint nedges = Y.size(0);
    uint nfeatures = X.size(1);

    TORCH_CHECK(X.requires_grad(), "X must require grad for invariant message passing backwards_kernel to be called.");
    TORCH_CHECK(Y.requires_grad(), "Y must require grad for invariant message passing backwards_kernel to be called.");
    TORCH_CHECK(radial.requires_grad(), "radial must require grad for invariant message passing backwards_kernel to be called.");

    torch::Tensor gradRadial = torch::empty_like(radial,
                                                 torch::TensorOptions()
                                                     .dtype(radial.dtype())
                                                     .device(radial.device()));

    torch::Tensor gradX = torch::empty_like(X,
                                            torch::TensorOptions()
                                                .dtype(X.dtype())
                                                .device(X.device()));

    torch::Tensor gradY = torch::empty_like(Y,
                                            torch::TensorOptions()
                                                .dtype(Y.dtype())
                                                .device(Y.device()));

    torch::Tensor sorted_sender_idx = torch::argsort(sender_list).to(torch::kInt32);

    // torch::Tensor sorted_sender_idx = torch::zeros_like(sender_list);
    torch::Tensor first_occurences_node = calculate_first_occurences_gpu_with_sort(sender_list, X.size(0), 64, sorted_sender_idx);

    AT_DISPATCH_FLOATING_TYPES(
        X.type(), "backward_gpu", ([&]
                                   {
        dim3 blockDim(NWARPS_PER_BLOCK * WARP_SIZE, 1, 1);
        dim3 gridDim(nnodes, 1);

        void *sptr = nullptr;
        size_t space = 0;

        shared_array<scalar_t>(16 * X.size(1), sptr, &space);
        shared_array<scalar_t>(2 * NWARPS_PER_BLOCK * 16, sptr, &space); // buffer_Y, buffer_dY

        void *sptr_node = nullptr;
        size_t space_node = 0;

        shared_array<scalar_t>(NWARPS_PER_BLOCK * WARP_SIZE, sptr_node, &space_node); 
        shared_array<int32_t>(1024, sptr_node, &space_node); 
        //shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr_node, &space_node); // buffer_Y, buffer_dY


        if (nfeatures == 96)
        {
            backward_edge_inv_tp_kernel<scalar_t, 4, 3><<<gridDim, blockDim, space>>>(
                X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                gradRadial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());


            backward_node_inv_tp_kernel<scalar_t, 4, 3><<<gridDim, blockDim, space_node>>>(
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                sorted_sender_idx.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences_node.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());

        }
        else if (nfeatures == 64)
        {
            backward_edge_inv_tp_kernel<scalar_t, 4, 2><<<gridDim, blockDim, space>>>(
                X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                gradRadial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());


            backward_node_inv_tp_kernel<scalar_t, 4, 2><<<gridDim, blockDim, space_node>>>(
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                sorted_sender_idx.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences_node.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
        }
        else if (nfeatures == 32)
        {
            backward_edge_inv_tp_kernel<scalar_t, 4, 1><<<gridDim, blockDim, space>>>(
                X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                gradRadial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());


           backward_node_inv_tp_kernel<scalar_t, 4, 1><<<gridDim, blockDim, space_node>>>(
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                sorted_sender_idx.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences_node.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
        }else {
                backward_edge_inv_tp_kernel<scalar_t, 4, 4><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    gradRadial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

               backward_node_inv_tp_kernel<scalar_t, 4, 4><<<gridDim, blockDim, space_node>>>(
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                sorted_sender_idx.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences_node.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
            } }));

    hipDeviceSynchronize();

    return {gradX, gradY, gradRadial};
}