#include "hip/hip_runtime.h"
#include <torch/script.h>
#include <iostream>
#include <cuda/barrier>

using namespace std;
using namespace torch::indexing;
using namespace torch::autograd;

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)

#define FULL_MASK 0xffffffff

template <class T>
__host__ __device__ T *shared_array(std::size_t n_elements, void *&ptr,
                                    std::size_t *space = nullptr) noexcept
{
    const std::uintptr_t inptr = reinterpret_cast<uintptr_t>(ptr);
    const std::uintptr_t end = inptr + n_elements * sizeof(T);
    if (space)
        *space += static_cast<std::size_t>(end - inptr);
    ptr = reinterpret_cast<void *>(end);
    return reinterpret_cast<T *>(inptr);
}

__host__ __device__ int32_t find_integer_divisor(int32_t x, int32_t bdim)
{
    return (x + bdim - 1) / bdim;
}

#define WARP_SIZE 32
#define NWARPS_PER_BLOCK 4

#define NEIGHBOUR_NEDGES_PER_BLOCK 512

/*
This function takes a sorted input sender_list, which maps each edge to a node by index, and finds the positions of first occurences

This is required by the CUDA code so we can send all calculations per-node to a single block.

the function loads NEIGHBOUR_NEDGES_PER_BLOCK + 1 elements into shared memory, and then loops through the buffer twice. Once for even boundaries, once for odd boundaries.
*/

__global__ void calculate_first_occurences_kernel(const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> receiver_list,
                                                  const int32_t *__restrict__ sort_idx,
                                                  bool use_sort,
                                                  torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> first_occurences)
{
    extern __shared__ char buffer[];
    size_t offset = 0;
    int32_t *smem = reinterpret_cast<int32_t *>(buffer + offset);

    int32_t block_start = blockIdx.x * NEIGHBOUR_NEDGES_PER_BLOCK;

    int32_t nedges = receiver_list.size(0);

    // load all elements of senderlist needed by block into shared memory
    for (int32_t i = threadIdx.x; i < NEIGHBOUR_NEDGES_PER_BLOCK + 1; i += blockDim.x)
    {
        int32_t idx = block_start + i;

        if (idx < nedges)
        {
            if (use_sort)
            {
                smem[i] = receiver_list[sort_idx[idx]];
            }
            else
            {
                smem[i] = receiver_list[idx];
            }
        }
    }

    __syncthreads();

    // deal with even boundaries
    for (int32_t i = 2 * threadIdx.x; i < NEIGHBOUR_NEDGES_PER_BLOCK; i += 2 * blockDim.x)
    {
        int32_t idx = block_start + i;

        if (idx + 1 < nedges)
        {
            int32_t loc1 = smem[i];
            int32_t loc2 = smem[i + 1];

            if (loc1 != loc2)
            {
                first_occurences[loc2] = idx + 1;
            }
        }
    }

    // deal with odd boundaries
    for (int32_t i = 2 * threadIdx.x + 1; i < NEIGHBOUR_NEDGES_PER_BLOCK + 1; i += 2 * blockDim.x)
    {
        int32_t idx = block_start + i;

        if (idx + 1 < nedges)
        {
            int32_t loc1 = smem[i];
            int32_t loc2 = smem[i + 1];

            if (loc1 != loc2)
            {
                first_occurences[loc2] = idx + 1;
            }
        }
    }

    // deal with 0th element specifically, so we dont need to use torch::zeros
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        first_occurences[0] = 0;
    }
}

torch::Tensor calculate_first_occurences_gpu(torch::Tensor receiver_list, int64_t natoms, int64_t nthreadx)
{
    torch::Tensor first_occurences = torch::empty(natoms,
                                                  torch::TensorOptions()
                                                      .dtype(receiver_list.dtype())
                                                      .device(receiver_list.device()));

    int32_t nbx = find_integer_divisor(receiver_list.size(0), NEIGHBOUR_NEDGES_PER_BLOCK);

    dim3 block_dim(nbx);

    dim3 grid_dim(nthreadx, 1, 1);

    size_t total_buff_size = 0;

    total_buff_size += (NEIGHBOUR_NEDGES_PER_BLOCK + 1) * sizeof(int32_t);

    calculate_first_occurences_kernel<<<block_dim, grid_dim, total_buff_size>>>(
        receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
        nullptr,
        false,
        first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>());

    hipDeviceSynchronize();

    return first_occurences;
}

torch::Tensor calculate_first_occurences_gpu_with_sort(torch::Tensor receiver_list, int64_t natoms, int64_t nthreadx, torch::Tensor sort_indices)
{
    torch::Tensor first_occurences = torch::empty(natoms,
                                                  torch::TensorOptions()
                                                      .dtype(receiver_list.dtype())
                                                      .device(receiver_list.device()));

    int32_t nbx = find_integer_divisor(receiver_list.size(0), NEIGHBOUR_NEDGES_PER_BLOCK);

    dim3 block_dim(nbx);

    dim3 grid_dim(nthreadx, 1, 1);

    size_t total_buff_size = 0;

    total_buff_size += (NEIGHBOUR_NEDGES_PER_BLOCK + 1) * sizeof(int32_t);

    if (sort_indices.defined() && sort_indices.numel() != 0)
    {
        calculate_first_occurences_kernel<<<block_dim, grid_dim, total_buff_size>>>(
            receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
            sort_indices.data_ptr<int32_t>(),
            true,
            first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>());
    }

    hipDeviceSynchronize();

    return first_occurences;
}

template <typename scalar_t, const int TM, const int TN>
__global__ __launch_bounds__(NWARPS_PER_BLOCK *WARP_SIZE) void forward_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> X, // [nnodes nchannels]
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> Y, // [nedges, (L+1)**2]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> radial,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> sender_list,      //
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> receiver_list,    // which index we need to sum a particular edge into -> monotonically increasing.
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> first_occurences, // the indexes in reciever_list which deliniate the set of edges per node.
    torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> output)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    double *buffer_out = shared_array<double>(16 * NWARPS_PER_BLOCK * WARP_SIZE, sptr, &space);
    // scalar_t *buffer_Y = shared_array<scalar_t>(16 * NWARPS_PER_BLOCK, sptr, &space);

    scalar_t regM[16] = {0.0};
    scalar_t regN[TN] = {0.0};
    scalar_t regWeights[4 * TN] = {0.0};
    double result[16 * TN] = {0.0};

    const uint threadCol = threadIdx.x % WARP_SIZE;
    const uint threadRow = threadIdx.x / WARP_SIZE;

    const uint N = X.size(1);
    const uint edge_start = first_occurences[blockIdx.x];
    const uint edge_end = (blockIdx.x == first_occurences.size(0) - 1) ? receiver_list.size(0) : first_occurences[blockIdx.x + 1];
    const uint node_index = receiver_list[edge_start];

    const uint N_start = blockIdx.y * TN * WARP_SIZE;

    for (int tid = threadIdx.x; tid < 16 * NWARPS_PER_BLOCK * WARP_SIZE; tid += blockDim.x)
    {
        buffer_out[tid] = 0.0;
    }

    __syncthreads();

    // check if this node has neighbours
    if (edge_end - edge_start == 0)
    {
        return;
    }

    int niter = find_integer_divisor(edge_end - edge_start, NWARPS_PER_BLOCK);

    for (uint ni = 0; ni < niter; ni++)
    {
        uint edge = edge_start + ni * NWARPS_PER_BLOCK + threadRow;

        if (edge < edge_end)
        {

            /*if (threadCol < 16)
            {
                buffer_Y[threadCol * NWARPS_PER_BLOCK + threadRow] = Y[edge][threadCol];
            } */

            __syncwarp();

            uint sender_id = sender_list[edge];

            for (uint n = 0; n < TN; n++)
            {
                if (N_start + n * WARP_SIZE + threadCol < N)
                    regN[n] = X[sender_id][N_start + n * WARP_SIZE + threadCol];
            }

            // load first into registers
            for (uint m = 0; m < 16; m++)
            {
                regM[m] = Y[edge][m];
            }

            for (uint n = 0; n < TN; n++)
            {
                if (N_start + n * WARP_SIZE + threadCol < N)
                {
                    for (int L = 0; L < 4; L++)
                    {
                        regWeights[L * TN + n] = radial[edge][L][n * WARP_SIZE + threadCol];
                    }
                }
            }

            for (uint m = 0; m < 16; m++)
            {
                int32_t lm_index = sqrt(m);
                // perform outer product in registers
                for (uint n = 0; n < TN; n++)
                {
                    if (N_start + n * WARP_SIZE + threadCol < N)
                    {

                        result[m * TN + n] += ((double)regWeights[lm_index * TN + n]) * ((double)regM[m]) * ((double)regN[n]);

                        // result[m * TN + n] += regWeights[lm_index * TN + n] * regM[m] * regN[n];
                    }
                }
            }
        }
    }

        // now need to accumulate partial results from each warp.
    for (int n = 0; n < TN; n++)
    {
        __syncthreads();
        
        for (int m = 0; m < 16; m++)
        {
            buffer_out[m * NWARPS_PER_BLOCK * WARP_SIZE + threadRow * WARP_SIZE + threadCol] = result[m * TN + n];
        }

        __syncthreads();

        if (N_start + n * WARP_SIZE + threadCol < N)
        {
            for (int m = threadRow; m < 16; m += NWARPS_PER_BLOCK)
            {
                double tmp = 0.0;

                for (int j = 0; j < NWARPS_PER_BLOCK; j++)
                {
                    tmp += buffer_out[m * NWARPS_PER_BLOCK * WARP_SIZE + j * WARP_SIZE + threadCol];
                }

                output[node_index][m][N_start + n * WARP_SIZE + threadCol] = tmp;
            }
        }
    }
}

torch::Tensor forward_gpu(
    torch::Tensor X,
    torch::Tensor Y,
    torch::Tensor radial,
    torch::Tensor sender_list,
    torch::Tensor receiver_list,
    torch::Tensor first_occurences,
    const int64_t nnodes)
{

    const uint nedges = Y.size(0);
    const int nspherical_harm = Y.size(1);
    const int nfeatures = X.size(1);

    TORCH_CHECK(nfeatures % WARP_SIZE == 0, "feature dimension must be a multiple of 32");
    TORCH_CHECK(nspherical_harm == 16, "number of edge spherical harmonics must be 16");
    TORCH_CHECK(nfeatures <= 128, "feature dimension cannot be greater than 128");

    torch::Tensor output = torch::empty({nnodes, nspherical_harm, nfeatures},
                                        torch::TensorOptions()
                                            .dtype(X.dtype())
                                            .device(X.device()));

    dim3 gridDim(nnodes, find_integer_divisor(nfeatures, 128));

    dim3 blockDim(NWARPS_PER_BLOCK * WARP_SIZE, 1, 1);

    AT_DISPATCH_FLOATING_TYPES(
        X.type(), "forward_gpu", ([&]
                                  {
            size_t space = 0;
            void *sptr = nullptr;

            shared_array<double>(16 * NWARPS_PER_BLOCK * WARP_SIZE, sptr, &space); // 64 * 128 * 8 = 65kb of shared memory...too large...
            //shared_array<scalar_t>(16 * NWARPS_PER_BLOCK, sptr, &space);
            if (nfeatures >= 128)
            {
                forward_kernel<scalar_t, 4, 4><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    output.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
            }
            else if (nfeatures == 96)
            {
                forward_kernel<scalar_t, 4, 3><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    output.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
            }else if (nfeatures == 64)
            {
                forward_kernel<scalar_t, 4, 2><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    output.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
            }else if (nfeatures == 32)
            {
                forward_kernel<scalar_t, 4, 1><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    output.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());
            } }

                                  ));

    hipDeviceSynchronize();

    return output;
}

template <typename scalar_t, const int TM, const int TN>
__global__ void __launch_bounds__(NWARPS_PER_BLOCK *WARP_SIZE) backward_edge_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> X,       // [nedges, feat]
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> Y,       // [nedges, m]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> radial,  // [nedges, LMAX, feat]
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> grad_in, // [nnodes, m, feat]
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> sender_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> receiver_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> first_occurences,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> gradY,
    torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> grad_radial)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    scalar_t *buffer_grad_in = shared_array<scalar_t>(16 * X.size(1), sptr, &space);
    scalar_t *buffer_Y = shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr, &space);
    scalar_t *buffer_dY = shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr, &space);

    const uint threadCol = threadIdx.x % WARP_SIZE;
    const uint threadRow = threadIdx.x / WARP_SIZE;

    const uint edge_start = first_occurences[blockIdx.x];
    const uint node_index = receiver_list[edge_start];
    const uint edge_end = (blockIdx.x == first_occurences.size(0) - 1) ? receiver_list.size(0) : first_occurences[blockIdx.x + 1];

    const uint N_start = blockIdx.y * TN * WARP_SIZE;

    scalar_t regX[TN] = {0.0};
    scalar_t regW[4 * TN] = {0.0};

    scalar_t regGradW[4 * TN] = {0.0};

    if (edge_end - edge_start == 0)
    {
        return;
    }

    for (int m = 0; m < 16 / NWARPS_PER_BLOCK; m++)
    {
        for (int n = 0; n < TN; n++)
        {
            // if (N_start + n * WARP_SIZE + threadCol < X.size(1))
            buffer_grad_in[(m * NWARPS_PER_BLOCK + threadRow) * X.size(1) + n * WARP_SIZE + threadCol] = grad_in[node_index][m * NWARPS_PER_BLOCK + threadRow][N_start + n * WARP_SIZE + threadCol];
        }
    }

    __syncthreads();

    int niter = find_integer_divisor(edge_end - edge_start, NWARPS_PER_BLOCK);

    for (uint ni = 0; ni < niter; ni++)
    {
        uint edge = edge_start + ni * NWARPS_PER_BLOCK + threadRow;

        if (edge < edge_end)
        {
            uint sender_id = sender_list[edge];

            if (threadCol < 16)
            {
                buffer_Y[threadCol * NWARPS_PER_BLOCK + threadRow] = Y[edge][threadCol];
                buffer_dY[threadCol * NWARPS_PER_BLOCK + threadRow] = 0.0;
            }

            __syncwarp();

            for (int n = 0; n < TN; n++)
            {

                regX[n] = X[sender_id][N_start + n * WARP_SIZE + threadCol];
            }

            for (int n = 0; n < TN; n++)
            {
                for (int L = 0; L < 4; L++)
                {
                    regGradW[L * TN + n] = 0.0;
                    regW[L * TN + n] = radial[edge][L][N_start + n * WARP_SIZE + threadCol];
                }
            }

            for (int L = 0; L < 4; L++)
            {
                uint mstart = L * L;
                uint mend = (L + 1) * (L + 1);

                for (int m = mstart; m < mend; m++)
                {
                    scalar_t sph = buffer_Y[m * NWARPS_PER_BLOCK + threadRow]; // Y[edge][m];

                    scalar_t dgradY = 0.0;

                    for (int n = 0; n < TN; n++)
                    {
                        //  scalar_t gradin = regGradIn[m * TN + n];
                        scalar_t gradin = buffer_grad_in[m * X.size(1) + n * WARP_SIZE + threadCol];
                        scalar_t w = regW[L * TN + n];

                        regGradW[L * TN + n] += sph * regX[n] * gradin;

                        dgradY += gradin * w * regX[n];
                    }

                    for (int offset = 16; offset > 0; offset /= 2)
                    {
                        dgradY += __shfl_down_sync(FULL_MASK, dgradY, offset, WARP_SIZE);
                    }

                    // threadIdx % WARP_SIZE = 0 dgradY contains the derivative of the output wrt. Y
                    if (threadCol == 0)
                        buffer_dY[m * NWARPS_PER_BLOCK + threadRow] = dgradY;
                }
            }
        }

        __syncthreads();

        if (edge < edge_end)
        {
            if (threadCol < 16)
            {
                gradY[edge][threadCol] = buffer_dY[threadCol * NWARPS_PER_BLOCK + threadRow];
            }

            for (int n = 0; n < TN; n++)
            {
                //  write grad_radial
                for (int L = 0; L < 4; L++)
                {
                    grad_radial[edge][L][N_start + n * WARP_SIZE + threadCol] = regGradW[L * TN + n];
                }
            }
        }
    }
}

template <typename scalar_t, const int TN>
__global__ void __launch_bounds__(NWARPS_PER_BLOCK *WARP_SIZE) backward_node_kernel(
    const torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> Y,
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> radial,
    const torch::PackedTensorAccessor64<scalar_t, 3, torch::RestrictPtrTraits> grad_in,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> sender_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> receiver_list,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> sorted_sender_idx,
    const torch::PackedTensorAccessor64<int32_t, 1, torch::RestrictPtrTraits> first_occurences,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits> gradX)
{

    extern __shared__ char buffer[];

    void *sptr = buffer;
    size_t space = 0;

    double *buffer_out = shared_array<double>(NWARPS_PER_BLOCK * WARP_SIZE, sptr, &space);
    scalar_t *buffer_Y = shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr, &space);

    const uint threadCol = threadIdx.x % WARP_SIZE;
    const uint threadRow = threadIdx.x / WARP_SIZE;

    const uint edge_start = first_occurences[blockIdx.x];
    const uint node_index = sender_list[sorted_sender_idx[edge_start]];
    const uint edge_end = (blockIdx.x == first_occurences.size(0) - 1) ? sender_list.size(0) : first_occurences[blockIdx.x + 1];

    const uint N_start = blockIdx.y * TN * WARP_SIZE;

    scalar_t regGradIn[16 * TN] = {0.0};
    scalar_t regW[4 * TN] = {0.0};
    double regGradX[TN] = {0.0};

    if (edge_end - edge_start == 0)
    {
        return;
    }

    int niter = find_integer_divisor(edge_end - edge_start, NWARPS_PER_BLOCK);

    for (uint ni = 0; ni < niter; ni++)
    {
        __syncthreads();

        uint edge = edge_start + ni * NWARPS_PER_BLOCK + threadRow;

        if (edge < edge_end)
        {
            uint sorted_id = sorted_sender_idx[edge];
            uint receiver_id = receiver_list[sorted_id];
            if (threadCol < 16)
            {
                buffer_Y[threadCol * NWARPS_PER_BLOCK + threadRow] = Y[sorted_id][threadCol];
            }
            __syncwarp();

            for (int m = 0; m < 16; m++)
            {
                for (int n = 0; n < TN; n++)
                {
                    regGradIn[m * TN + n] = grad_in[receiver_id][m][N_start + n * WARP_SIZE + threadCol];
                }
            }

            for (int n = 0; n < TN; n++)
            {
                for (int L = 0; L < 4; L++)
                {
                    regW[L * TN + n] = radial[sorted_id][L][N_start + n * WARP_SIZE + threadCol];
                }
            }

            for (int m = 0; m < 16; m++)
            {
                int L = sqrt(m);
                for (int n = 0; n < TN; n++)
                {
                    regGradX[n] += ((double)regW[L * TN + n]) * ((double)regGradIn[m * TN + n]) * ((double)buffer_Y[m * NWARPS_PER_BLOCK + threadRow]);
                }
            }
        }
    }

    // now need to accumulate partial results from each warp.
    for (int n = 0; n < TN; n++)
    {
        __syncthreads();

        buffer_out[threadRow * WARP_SIZE + threadCol] = regGradX[n];

        __syncthreads();

        if (threadRow == 0)
        {

            double tmp = 0.0;

            for (int j = 0; j < NWARPS_PER_BLOCK; j++)
            {
                tmp += buffer_out[j * WARP_SIZE + threadCol];
            }

            gradX[node_index][N_start + n * WARP_SIZE + threadCol] = tmp;
        }
    }
}

std::vector<torch::Tensor> backward_gpu(torch::Tensor X,
                                        torch::Tensor Y,
                                        torch::Tensor radial,
                                        torch::Tensor grad_in,
                                        torch::Tensor sender_list,
                                        torch::Tensor receiver_list,
                                        torch::Tensor first_occurences,
                                        const int64_t nnodes)
{
    uint nedges = Y.size(0);
    uint nfeatures = X.size(1);

    TORCH_CHECK(X.requires_grad(), "X must require grad for invariant message passing backwards_kernel to be called.");
    TORCH_CHECK(Y.requires_grad(), "Y must require grad for invariant message passing backwards_kernel to be called.");
    TORCH_CHECK(radial.requires_grad(), "radial must require grad for invariant message passing backwards_kernel to be called.");

    torch::Tensor gradRadial = torch::empty_like(radial,
                                                 torch::TensorOptions()
                                                     .dtype(radial.dtype())
                                                     .device(radial.device()));

    torch::Tensor gradX = torch::empty({X.size(0), X.size(1)},
                                       torch::TensorOptions()
                                           .dtype(X.dtype())
                                           .device(X.device()));

    torch::Tensor gradY = torch::empty_like(Y,
                                            torch::TensorOptions()
                                                .dtype(Y.dtype())
                                                .device(Y.device()));

    torch::Tensor sorted_sender_idx = torch::argsort(sender_list).to(torch::kInt32);
    torch::Tensor first_occurences_node = calculate_first_occurences_gpu_with_sort(sender_list, X.size(0), 64, sorted_sender_idx);

    AT_DISPATCH_FLOATING_TYPES(
        X.type(), "backward_gpu", ([&]
                                   {
        dim3 blockDim(NWARPS_PER_BLOCK * WARP_SIZE, 1, 1);
        dim3 gridDim(nnodes, 1);

        void *sptr = nullptr;
        size_t space = 0;

        shared_array<scalar_t>(16 * X.size(1), sptr, &space);
        shared_array<scalar_t>(2 * NWARPS_PER_BLOCK * 16, sptr, &space); // buffer_Y, buffer_dY

        void *sptr_node = nullptr;
        size_t space_node = 0;

        shared_array<double>(NWARPS_PER_BLOCK * WARP_SIZE, sptr_node, &space_node); 
        shared_array<scalar_t>(NWARPS_PER_BLOCK * 16, sptr_node, &space_node); // buffer_Y, buffer_dY


        if (nfeatures == 96)
        {

            backward_edge_kernel<scalar_t, 4, 3><<<gridDim, blockDim, space>>>(
                X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                gradRadial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

            hipDeviceSynchronize();

            backward_node_kernel<scalar_t, 3><<<gridDim, blockDim, space_node>>>(
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                sorted_sender_idx.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences_node.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());

        }
        else if (nfeatures == 64)
        {
            backward_edge_kernel<scalar_t, 4, 2><<<gridDim, blockDim, space>>>(
                X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                gradRadial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

            hipDeviceSynchronize();

            backward_node_kernel<scalar_t, 2><<<gridDim, blockDim, space_node>>>(
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                sorted_sender_idx.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences_node.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
        }
        else if (nfeatures == 32)
        {
            backward_edge_kernel<scalar_t, 4, 1><<<gridDim, blockDim, space>>>(
                X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                gradRadial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

            hipDeviceSynchronize();

           backward_node_kernel<scalar_t, 1><<<gridDim, blockDim, space_node>>>(
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                sorted_sender_idx.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences_node.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
        }else {
                backward_edge_kernel<scalar_t, 4, 4><<<gridDim, blockDim, space>>>(
                    X.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                    sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    first_occurences.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                    gradY.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                    gradRadial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>());

                hipDeviceSynchronize();

               backward_node_kernel<scalar_t, 4><<<gridDim, blockDim, space_node>>>(
                Y.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>(),
                radial.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                grad_in.packed_accessor64<scalar_t, 3, torch::RestrictPtrTraits>(),
                sender_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                receiver_list.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                sorted_sender_idx.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                first_occurences_node.packed_accessor64<int32_t, 1, torch::RestrictPtrTraits>(),
                gradX.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
            } }));

    hipDeviceSynchronize();

    return {gradX, gradY, gradRadial};
}

class InvariantMessagePassingTPAutograd : public Function<InvariantMessagePassingTPAutograd>
{
public:
    static torch::Tensor forward(
        AutogradContext *ctx,
        torch::Tensor X,
        torch::Tensor Y,
        torch::Tensor radial,
        torch::Tensor sender_list,
        torch::Tensor receiver_list,
        const int64_t nnodes)
    {

        torch::Tensor first_occurences = calculate_first_occurences_gpu(receiver_list, nnodes, 64);

        if (X.requires_grad() || Y.requires_grad() || radial.requires_grad())
        {
            ctx->saved_data["nnodes"] = nnodes;
            ctx->save_for_backward({X, Y, radial, sender_list, receiver_list, first_occurences});
        }

        return forward_gpu(X, Y, radial, sender_list, receiver_list, first_occurences, nnodes);
    }

    static variable_list backward(AutogradContext *ctx, variable_list grad_outputs)
    {

        auto saved_variables = ctx->get_saved_variables();

        auto X = saved_variables[0];
        auto Y = saved_variables[1];
        auto radial = saved_variables[2];
        auto sender_list = saved_variables[3];
        auto receiver_list = saved_variables[4];
        auto first_occurences = saved_variables[5];

        int64_t nnodes = ctx->saved_data["nnodes"].toInt();

        auto result = backward_gpu(X, Y, radial, grad_outputs[0], sender_list, receiver_list, first_occurences, nnodes);

        torch::Tensor undef;

        return {result[0], result[1], result[2], undef, undef, undef};
    }
};

torch::Tensor invariant_message_passing_tensor_product(
    torch::Tensor X,
    torch::Tensor Y,
    torch::Tensor radial,
    torch::Tensor sender_list,
    torch::Tensor receiver_list,
    const int64_t nnodes)
{
    return InvariantMessagePassingTPAutograd::apply(X, Y, radial, sender_list, receiver_list, nnodes);
}

TORCH_LIBRARY(invariant_tp_new, m)
{
    m.def("forward", &invariant_message_passing_tensor_product);
    m.def("calculate_first_occurences", &calculate_first_occurences_gpu);
    m.def("calculate_first_occurences_with_sort", &calculate_first_occurences_gpu_with_sort);
}
